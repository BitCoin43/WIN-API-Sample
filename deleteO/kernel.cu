﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <ctime>

const int size = 1080 * 1920 * 1024;

__global__ void test(int* input, int* output, int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int g = input[x] * 8231 + 3224;
	g /= 3;
	output[x] = g;
}

__global__ void set(int* input, int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = 0; i < 1920; i++) {
		input[x + i * 1080 * 1024] = x;
	}
	
}

uint64_t now() {
	return std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
}

int main() {
	int* inp = nullptr;
	int* out = nullptr;

	uint64_t befor = now();

	hipMalloc((void**)&inp, sizeof(int) * size);
	hipMalloc((void**)&out, sizeof(int) * size);

	set<<<1080, 1024>>>(inp, size);


	
	hipFree(inp);
	hipFree(out);

	uint64_t after = now();
	std::cout << "For all: " << after - befor << "ms" << std::endl;

	return 0;
}