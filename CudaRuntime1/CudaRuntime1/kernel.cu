#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

__device__ int f(int n) {
    if (n == 0) return 5;
    if (n > 0 && n % 2 == 0) return 1 + f(n / 2);
    return f(n / 2);
}

__global__ void g(int *c)
{
    int i = blockIdx.x;
    int y = f(i);
    
    if (y == 7)
        *c = *c + 1;
  
}

int main()
{
    int* c = nullptr;
    int h = 0;

    hipMalloc((void**)&c, sizeof(int));
    g << <1000000000, 1 >> > (c);
    
    hipMemcpy(&h, c, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << h;
    return 0;
}

